#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/norm_select_layer.hpp"

namespace caffe {
	
template <typename Dtype>
__global__ void kernel_channel_sum(const int num, const int channels,const int spatial_dim, Dtype epsilon,
								const Dtype* data, Dtype* norm_data) {
	CUDA_KERNEL_LOOP(index, num * spatial_dim) {
		int n = index / spatial_dim; 
		int s = index % spatial_dim; 
		Dtype sum = 0; 
		for (int c = 0; c < channels; ++c) {
			sum += data[(n * channels + c) * spatial_dim + s]; 
		}
		norm_data[index] = sum + epsilon; 
	}
}

template <typename Dtype>
__global__ void kernel_norm_select(const int num, const int channels,const int spatial_dim,
								const Dtype* norm_data,Dtype* data,const Dtype threshold) {
    CUDA_KERNEL_LOOP(index,num * spatial_dim) {
		int n = index / spatial_dim; 
		int s = index % spatial_dim; 
		data[n] = (norm_data[n*spatial_dim+s] < threshold)? Dtype(0) : Dtype(1); 
	}
}

template <typename Dtype>
__global__ void kernel_forward(const int num,const int channels,const int spatial_dim,
								const Dtype* norm_data,const Dtype* bottom_data,Dtype* top_data,const Dtype threshold){
	CUDA_KERNEL_LOOP(index,num*spatial_dim) {
		int n = index / spatial_dim; 
		int s = index % spatial_dim;
		if (norm_data[n*spatial_dim + s] < threshold)
		{
			for (int c = 0; c < channels; ++c) 
				top_data[(n*channels+c)*spatial_dim + s] = Dtype(0); 
		}
		else{
			for (int c = 0; c < channels; ++c) 
				top_data[(n*channels+c)*spatial_dim + s] = bottom_data[(n*channels+c)*spatial_dim + s]; 			
		}
	}
}

template <typename Dtype>
__global__ void kernel_backward(const int num,const int channels,const int spatial_dim,
						const Dtype* norm_data,const Dtype* top_diff,Dtype* bottom_diff,const Dtype threshold){
	CUDA_KERNEL_LOOP(index,num*spatial_dim){
		int n = index / spatial_dim; 
		int s = index % spatial_dim; 
		if (norm_data[n*spatial_dim + s] < threshold)
		{
			for (int c = 0; c < channels; ++c)
				bottom_diff[(n*channels+c)*spatial_dim + s] = Dtype(0); 
		}
		else 
		{
			for (int c = 0; c < channels; ++c)
				bottom_diff[(n*channels+c)*spatial_dim + s] = top_diff[(n*channels+c)*spatial_dim + s]; 
		}
	}
}

template <typename Dtype>
void NormSelectLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
	const vector<Blob<Dtype>*>& top) {
	
	const Dtype* bottom_data = bottom[0]->gpu_data(); 
	Dtype* top_data = top[0]->mutable_gpu_data(); 
	Dtype* square_data = squared_.mutable_gpu_data(); 
	Dtype* norm_data = norm_.mutable_gpu_data(); 
	
	int num = bottom[0]->num(); 
	int channels = bottom[0]->channels(); 
	int spatial_dim = bottom[0]->height() * bottom[0]->width(); 
	
	caffe_gpu_powx(num*channels*spatial_dim, bottom_data, Dtype(2),square_data); 
	kernel_channel_sum<Dtype> << <CAFFE_GET_BLOCKS(num*spatial_dim),
		CAFFE_CUDA_NUM_THREADS >> >(num, channels, spatial_dim, 1e-12, square_data, norm_data); 
	caffe_gpu_powx(num * spatial_dim, norm_data, Dtype(-0.5), norm_data);
	kernel_norm_select<Dtype> << <CAFFE_GET_BLOCKS(num*spatial_dim),
		CAFFE_CUDA_NUM_THREADS >> >(num, channels, spatial_dim, norm_data, top[1]->mutable_gpu_data(),threshold_); 
	kernel_forward<Dtype> << <CAFFE_GET_BLOCKS(num*spatial_dim),
		CAFFE_CUDA_NUM_THREADS >> >(num,channels,spatial_dim,norm_data,bottom_data,top_data,threshold_);
	
}

template <typename Dtype>
void NormSelectLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
	const vector<bool>& propagate_down,const vector<Blob<Dtype>*>& bottom) {
	
	const Dtype* top_diff = top[0]->gpu_diff(); 
	const Dtype* norm_data = norm_.gpu_data(); 
	Dtype* bottom_diff = bottom[0]->mutable_gpu_diff(); 
	
	int num = top[0]->num(); 
	int channels = top[0]->channels(); 
	int spatial_dim = bottom[0]->height() * bottom[0]->width(); 
	
	kernel_backward<Dtype> << <CAFFE_GET_BLOCKS(num*spatial_dim),
		CAFFE_CUDA_NUM_THREADS >> >(num,channels,spatial_dim,norm_data,top_diff,bottom_diff,threshold_); 
}

INSTANTIATE_LAYER_GPU_FUNCS(NormSelectLayer); 

} // namespace caffe
